#include "hip/hip_runtime.h"
#include "spring_attacher_gpu.hpp"
#include <sofa/gpu/cuda/CudaCommon.h>
#include <iostream>
#include <cuda_gl_interop.h>

#include "kernel_math.hpp"

struct assoc_t{
	int prim;
	float3 bary;
	bool valid;

	__device__
	bool is_valid()
	{
		return valid;
	}
};

__device__ 
assoc_t unpack_association_data(float4 &packed)
{
	assoc_t assoc;

	//for(int i=0; i<3; ++i) get(assoc.bary,i) = get(packed,i);
	assoc.bary.x = packed.x;
	assoc.bary.y = packed.y;
	assoc.bary.z = packed.z;
	if(packed.w >= 0){
		assoc.prim = (unsigned int) packed.w;
		assoc.valid = true;
	} else {
		assoc.prim = 0;
		assoc.valid = false;
	}

	return assoc;
}

__device__
int get_index_x()
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	return index;
}

__device__
int2 get_index_xy()
{
	int2 index;
	index.x = blockDim.x * blockIdx.x + threadIdx.x;
	index.y = blockDim.y * blockIdx.y + threadIdx.y;
	return index;
}

__device__
void fetch_vertices(float3* vertices, const float3* vertex_list, uint3 &vertex_indices)
{
	for(size_t i=0; i<3; ++i) vertices[i] = vertex_list[get(vertex_indices,i)];
}

__global__
void clear_kernel(float3* anchor_list, float* weights, unsigned int size)
{
	int index = get_index_x();

	if(index < size) {
		anchor_list[index].x = 0;
		anchor_list[index].y = 0;
		anchor_list[index].z = 0;
		weights[index] = 0; 
	}
}

__global__
void accumulate_forces_kernel(float3* data, float4* association_data, float3* anchor_list, uint3* triangles,
		float3* vertex_list, float3* normal_list, float* weights,
		unsigned int width, unsigned int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if( x >= width || y >= height) return;

	float3 data_point = data[x+y*width];

	if(!isfinite(data_point)) return;

	assoc_t assoc = unpack_association_data(association_data[x+y*width]);

	if(!assoc.valid) {
		for(int dx=-2; dx<=2; ++dx) {
			for(int dy=-2; dy <=2; ++dy) {
				int idx = (x+dx)+(y+dy)*width;
				assoc = unpack_association_data(association_data[idx]);
				if(assoc.valid) break;
			}
		}
	}

	if(!assoc.valid) return;

	uint3 triangle = triangles[assoc.prim];

	for(int i=0; i<3; ++i)
	{
		unsigned int vertex_index = get(triangle,i);
		float3 vertex = vertex_list[vertex_index];
		float3 normal = normal_list[vertex_index];
		normalize(normal);
		float3 &anchor = anchor_list[vertex_index];
		float &weight = weights[vertex_index];

		float3 relative = data_point - vertex;

		float len = norm(relative);
		
		if(len < 25) {
			float l = relative*normal;
			if(l > 5) l = 5;
			float3 projected = normal * l;

			if(len>5) projected *= 25/(len*len);

			if(isfinite(projected)){
				accumulate(anchor, projected);
				atomicAdd(&weight ,1);
			}
		}
	}
}

__global__
void attach_kernel(float3* anchors, float3* vertices, float* weights, unsigned int vertex_count)
{
	int index = get_index_x();
	if(index >= vertex_count) return;

	float3 &anchor = anchors[index];
	const float3 &vertex = vertices[index];
	const float &weight = weights[index];

	float3 force = anchor;
	float3 displacement_vector = {0.0001f,0,0};

	if(weight > 0 && isfinite(force)) {
		displacement_vector = force / weight;
	}

	if(norm2(displacement_vector) < 0.00001f) {
		displacement_vector.x = 0.0001f;
		displacement_vector.y = 0;
		displacement_vector.z = 0;
	}
	anchor = vertex + displacement_vector;
}


void attach_gpu_springs(
		float *data,
		float *association_data,
		float *anchors,
		unsigned int *triangles,
		float *vertices,
		float *normals,
		float* weights,
		unsigned int vertex_count,
		unsigned int width,
		unsigned int height)
{
	dim3 vertex_dim = dim3(vertex_count,1,1);
	dim3 num_threads_vertices = dim3(BSIZE,1,1);
	dim3 num_blocks_vertices = calculate_block_count(vertex_dim, num_threads_vertices);

	dim3 image_dim = dim3(width,height,1);
	dim3 num_threads_image = dim3(8,8,1);
	dim3 num_blocks_image = calculate_block_count(image_dim, num_threads_image);


	clear_kernel<<<num_blocks_vertices,num_threads_vertices>>>(
			(float3*) anchors,
			weights,
			vertex_count
	);
	hipDeviceSynchronize();
	accumulate_forces_kernel<<<num_blocks_image, num_threads_image>>>(
			(float3*) data,
			(float4*) association_data,
			(float3*) anchors,
			(uint3*) triangles,
			(float3*) vertices,
			(float3*) normals,
			weights,
			width,
			height
	);
	hipDeviceSynchronize();

	attach_kernel<<<num_blocks_vertices, num_threads_vertices>>>(
		(float3*) anchors,
		(float3*) vertices,
		weights,
		vertex_count
	);
	hipDeviceSynchronize();
}

void map_association_surface(hipArray_t &array)
{
}
